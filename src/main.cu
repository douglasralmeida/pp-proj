/*
Projeto de Algoritmos Paralelos
*/


#include <hip/hip_runtime.h>
#include <limits.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime_api.h>

__global__ void cuda_olamundo(){
  printf("Ola Mundo direto da GPU!\n");
}

int main(int argc, const char * argv[]){
    int count;
    hipError_t erro;
    hipDeviceProp_t deviceProp; 
    
    cuda_olamundo<<<1,1>>>();
    
    hipProfilerStart();
    erro = hipGetDeviceCount(&count);
    if (erro != hipSuccess) {
        printf("Error: %s\n", hipGetErrorString(erro));
        exit(-1);
    }
    printf("Number of devices: %d\n", count);
    erro = hipGetDeviceProperties(&deviceProp, 0);
    if (erro != hipSuccess) {
        printf("Error: %s\n", hipGetErrorString(erro));
        exit(-1);
    }
    printf("\nDevice %d has compute capability %d.%d.\n", 0, deviceProp.major, deviceProp.minor);
    exit(EXIT_SUCCESS);
}
