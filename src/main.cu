/*
Projeto de Algoritmos Paralelos
*/


#include <hip/hip_runtime.h>
#include <limits.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

__global__ void cuda_olamundo(){
    printf("Ola Mundo direto da GPU!\n");
}

int main(int argc, const char * argv[]){
    cuda_olamundo<<<1,1>>>();
    exit(EXIT_SUCCESS);
}