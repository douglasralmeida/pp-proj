/*
Projeto de Algoritmos Paralelos
*/


#include <hip/hip_runtime.h>
#include <limits.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime_api.h>

__global__ void cuda_olamundo(){
  printf("Ola Mundo direto da GPU!\n");
}

int main(int argc, const char * argv[]){
    int count;
    hipError_t erro;
    hipDeviceProp_t deviceProp; 
    
    hipProfilerStart();
    erro = hipGetDeviceCount(&count);
    if (erro != hipSuccess) {
        printf("Erro: %s\n", hipGetErrorString(erro));
        exit(-1);
    }
    printf("Numero de dispositivos: %d\n", count);
    erro = hipGetDeviceProperties(&deviceProp, 0);
    if (erro != hipSuccess) {
        printf("Erro: %s\n", hipGetErrorString(erro));
        exit(-1);
    }
    printf("Dispositivo %d tem a capacidade computacional %d.%d.\n\n", 0, deviceProp.major, deviceProp.minor);
    
    cuda_olamundo<<<1,1>>>();
    hipDeviceReset();
    exit(EXIT_SUCCESS);
}
