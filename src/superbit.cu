#include "hip/hip_runtime.h"
/*
** Projeto de Algoritmos Paralelos
** Algoritmo Superbit - Implementação em CPU
*/

#include <cmath>
#include <random>
#include <ctime>
#include "array.hpp"
#include "math.hpp"
#include "superbit.hpp"

#define THREADS_PER_BLOCK 512

__global__ void cudaComputeSignature(double* hyperplanes, double* v, int* dimensions, bool* sig, long* hyperp_length) {
    int d_dimensions = *dimensions;
    long pos = (threadIdx.x + blockDim.x * blockIdx.x) * d_dimensions;
    double sum = 0.0;

    for (int i = 0; i < d_dimensions; i++)
        sum += hyperplanes[i+pos] * v[i];
    sig[threadIdx.x + blockDim.x * blockIdx.x] = (sum>=0);
}

Superbit::Superbit(const int _dimensions, int _superbit, long _length, int _seed):dimensions(_dimensions) {
    hpbuilder_t builderdata;

    if (_superbit < 1 || _superbit > _dimensions) {
        cout << "Erro. Profundidade N do SuperBit deve estar entre 1 e o número de dimensões." << endl;
        exit(EXIT_FAILURE);
    }
    hyperp_length = _superbit * _length;
    hyperplanes = Array::alloc2d(hyperp_length, dimensions);
    builderdata.v = Array::alloc2d(hyperp_length, dimensions);
    builderdata.superbit = _superbit;
    builderdata.length = _length;
    builderdata.seed = _seed;
    builderdata.w = hyperplanes;
    buildHyperplanes(&builderdata);
    Array::dealloc2d(&builderdata.v);

    hipMalloc(&d_v,sizeof(double)*dimensions);
    hipMalloc(&d_sig,sizeof(bool)*hyperp_length);
    hipMalloc(&d_hyperplanes,sizeof(double)*(hyperp_length*dimensions));
    hipMalloc(&d_hyperp_length,sizeof(long));
    hipMalloc(&d_dimensions,sizeof(int));
    hipMemcpy(d_hyperp_length,&hyperp_length,sizeof(long),hipMemcpyHostToDevice);
    hipMemcpy(d_dimensions,&dimensions,sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(d_hyperplanes,hyperplanes,sizeof(double)*(hyperp_length * dimensions),hipMemcpyHostToDevice);
}

Superbit::Superbit(const int _dimensions, int _superbit, long _length):
    Superbit(_dimensions, _superbit, _length, time(NULL)) {}

Superbit::~Superbit() {
    Array::dealloc2d(&hyperplanes);
    hipFree(d_V);
    hipFree(d_sig);
    hipFree(d_hyperplanes);
    hipFree(d_hyperp_length);
    hipFree(d_dimensions);
}

void Superbit::buildHyperplanes(hpbuilder_t *builderdata) {
    long i, j, k;
    std::default_random_engine generator(builderdata->seed);
    std::normal_distribution<long double> distribution(0.0, 1.0);
    double* v = builderdata->v;
    double* w = builderdata->w;

    for (i = 0; i < hyperp_length; i++) {
        long x = i * dimensions;
        for (j = 0; j < dimensions; j++)
            v[x + j] = distribution(generator);
        Math::normalize(v + x, dimensions);
    }

    for (i = 0; i <= (builderdata->length-1); i++) {
        for (j = 1; j <= builderdata->superbit; j++) {
            int pos = (i * builderdata->superbit + j - 1)*dimensions;
            Array::copy(v + pos, w + pos, dimensions);
            for (k = 1; k <= (j-1); k++) {
                int wpos = (i * builderdata->superbit + k - 1)*dimensions;
                Math::sub(w + pos,
                    Math::product(
                        Math::dotProduct(w + wpos, v + pos, dimensions),
                        w + wpos,
                        dimensions),
                    dimensions);
            }
            Math::normalize(w + pos, dimensions);
        }
    }
}

bool* Superbit::computeSignature(double* v) {
    bool* sig = new bool[hyperp_length];
    bool* sigaux = new bool[hyperp_length];
    
    int NUM_OF_BLOCKS = (hyperp_length + THREADS_PER_BLOCK - 1)/THREADS_PER_BLOCK;

    hipMemcpy(d_v,v,sizeof(double)*dimensions,hipMemcpyHostToDevice);
    cudaComputeSignature<<<NUM_OF_BLOCKS, THREADS_PER_BLOCK>>>(d_hyperplanes, d_v, d_dimensions, d_sig, d_hyperp_length);
    hipMemcpy(sig,d_sig,sizeof(bool)*hyperp_length,hipMemcpyDeviceToHost);

    return sig;
}

long Superbit::getSignatureLength() {
    return hyperp_length;
}

double Superbit::similarity(bool* s1, bool* s2) {
    double sum = 0;

    for (long i = 0; i < hyperp_length; i++)
        if (s1[i] == s2[i])
            sum++;
    sum /= hyperp_length;

    return cos((1 - sum) * acos(-1)); //acos(-1)=pi
}