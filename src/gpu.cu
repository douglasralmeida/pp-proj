/*
** Projeto de Algoritmos Paralelos
** Objeto GPU
*/

#include <iostream>
#include <hip/hip_runtime_api.h>
#include "gpu.hpp"

using namespace std;

//GPU com capacidade computacional 5.2
#define TAM_BLOCO 32

GPU::GPU() {
    int count;
    hipError_t error;

    cout << "Inicializando GPU..." << endl;
    hipProfilerStart();
    error = hipGetDeviceCount(&count);
    if (error != hipSuccess)
        exitWithFailure(error);
    cout << "Numero de dispositivos disponíveis: " << count << endl;
    error = hipGetDeviceProperties(&deviceProp, 0);
    if (error != hipSuccess) 
        exitWithFailure(error);
    cout << "Dispositivo 0 tem a capacidade computacional " << deviceProp.major << '.' << deviceProp.minor << endl;
}

GPU::~GPU() {
    hipDeviceReset();
}

void GPU::exitWithFailure(hipError_t error) {
    cout << "Erro de processamento na GPU: " << hipGetErrorString(error) << endl;
    exit(EXIT_FAILURE);
}
