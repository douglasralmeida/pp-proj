/*
** Projeto de Algoritmos Paralelos
** Objeto GPU
*/

#include <iostream>
#include <hip/hip_runtime_api.h>
#include "gpu.hpp"

using namespace std;

//GPU com capacidade computacional 5.2
#define TAM_BLOCO 32

GPU::GPU(bool silenceMode) {
    int count;
    hipError_t error;

    if (!silenceMode)
        cout << "Inicializando GPU..." << endl;
    hipProfilerStart();
    error = hipGetDeviceCount(&count);
    if (error != hipSuccess)
        exitWithFailure(error);
    if (!silenceMode)
        cout << "Numero de dispositivos disponíveis: " << count << endl;
    error = hipGetDeviceProperties(&deviceProp, 0);
    if (error != hipSuccess) 
        exitWithFailure(error);
    if (!silenceMode)
        cout << "Dispositivo 0 tem a capacidade computacional " << deviceProp.major << '.' << deviceProp.minor << endl;
}

GPU::~GPU() {
    hipDeviceReset();
}

void GPU::exitWithFailure(hipError_t error) {
    cout << "Erro de processamento na GPU: " << hipGetErrorString(error) << endl;
    exit(EXIT_FAILURE);
}
