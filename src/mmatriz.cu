#include "hip/hip_runtime.h"
/*
** Projeto de Algoritmos Paralelos
** Multiplicação de Matrizes
*/

#include <limits.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>
#include <hip/hip_runtime_api.h>

#define TAM_BLOCO 16

__global__ void cuda_multiplicarmatriz(float* M, float* N, float* R, int tamM, int tamN) {

    //índice do bloco
    int bx = blockIdx.x;
    int by = blockIdx.y; 
 
    // índice da thread
    int tx = threadIdx.x;
    int ty = threadIdx.y; 

    // índice da primeira submatriz de M processado pelo bloco
    int mComeco = tamM * TAM_BLOCO * by; 
 
    // índice da última submatriz de M processada pelo bloco
    int mFim   = mComeco + tamM - 1;
 
    // Tamanho do passo utilizado para interar através das submatrizes de M
    int mPasso  = TAM_BLOCO; 
 
    // Índice da primeira submatriz de N processada pelo bloco
    int nComeco = TAM_BLOCO * bx; 
 
    // Tamanho do passo utilizado para interar através das submatrizes de N
    int nPasso  = TAM_BLOCO * tamN;
 
    // O elemento computado pela thread
    float rRes = 0; 

    // Varre por todas as submatrizes de M e N requeridas
    // para computar o bloco de submatriz
    for (int m = mComeco, n = nComeco; m <= mFim; m += mPasso, n += nPasso) { 
 
        // Memoria compartilhada para a submatriz de M
        __shared__ float Msub[TAM_BLOCO][TAM_BLOCO]; 
 
        // Memoria compartilhada para a submatriz de N
        __shared__ float Nsub[TAM_BLOCO][TAM_BLOCO]; 
 
        // Carrega as matrizes da memória global para a memória
        // compartilhada. Cada thread carreg um elemento de cada
        // matriz
        Msub[ty][tx] = M[m + tamM * ty + tx];
        Nsub[ty][tx] = N[n + tamN * ty + tx];
 
        // Sincroniza para garantir que todas as matrizes foram
        // carregadas
        __syncthreads();
 
        // Multiplica as duas matrizes.
        // Cada thread computa um elemento
        // do bloco da submatriz
        for (int i = 0; i < TAM_BLOCO; ++i)
            rRes += Msub[ty][i] * Nsub[i][tx];
        
        // Sincroniza para grantir que a computação de multiplicação
        // está feita antes de carregar duas novas submatrizes de 
        // M e N na próxima interação
        __syncthreads();
    }
     // Esscre o bloco da sumatriz na memória global
     // Cada thread escreve  um único elemento
     int r = tamN * TAM_BLOCO * by + TAM_BLOCO * bx;
     R[r + tamN * ty + tx] = rRes;
}

// Função para rodar na CPU
// Computa R = M * N
//   aM é a altura de M
//   lM é a largura de M
//   lN é a largura de N
void multiplicar(const float* M, const float* N, float* R, int aM, int lM, int lN) {
    int tam;
 
    // Carrega M e N para a GPU
    float* Md;
    tam = aM * lM * sizeof(float);
    hipMalloc((void**)&Md, tam);
    hipMemcpy(Md, M, tam, hipMemcpyHostToDevice);
    float* Nd;
    tam = lM * lN * sizeof(float);
    hipMalloc((void**)&Nd, tam);
    hipMemcpy(Nd, N, tam, hipMemcpyHostToDevice); 
 
    // Aloca R na GPU
    float* Rd;
    tam = aM * lN * sizeof(float);
    hipMalloc((void**)&Rd, tam); 
 
    // Computa a configuração da execução assumindo que
    // as dimensões das matrizes são múltiplos de TAM_BLOCO
    dim3 dimBlock(TAM_BLOCO, TAM_BLOCO);
    dim3 dimGrid(lN / dimBlock.x, aM / dimBlock.y); 
 
    // Processa a computação na GPU
    cuda_multiplicarmatriz<<<dimGrid, dimBlock>>>(Md, Nd, lM, lN, Rd);
 
    // Carrega R da GPU
    hipMemcpy(R, Rd, tam, hipMemcpyDeviceToHost);  
 
    // Limpa a memória da GPU
    hipFree(Md);
    hipFree(Nd);
    hipFree(Rd); 
} 

int checkGpu() {
    int count;
    hipError_t erro;
    
    hipProfilerStart();
    erro = hipGetDeviceCount(&count);
    if (erro != hipSuccess) {
        printf("Erro: %s\n", hipGetErrorString(erro));
        return 0;
    }
    if (count < 1) {
        printf("Erro: %s\n", "Este computador não possui um dispositivo com GPU compatível com CUDA disponível.");
        return 0;
    }

    return 1;
}

void matriz_preencher(float* A, int tam) {
    for (int i = 0; i < tam*tam; i++)
        A[i] = rand() % 100;
}

void matriz_exibir(float* A, int tam) {
    for (int i = 0; i < tam; i++) {
        for (int j = 0; j < tam; j++) {
            printf("%d ", A[tam*i+j]);
        }
        printf("\n");
    }
}

int main(int argc, const char * argv[]){
    float A[TAM_BLOCO*TAM_BLOCO];
    float B[TAM_BLOCO*TAM_BLOCO];
    float C[TAM_BLOCO*TAM_BLOCO];
    int aA = TAM_BLOCO;
    int lA = TAM_BLOCO;
    int lB = TAM_BLOCO;

    if (!checkGpu())
        exit(EXIT_FAILURE);
    srand(time(NULL));

    matriz_preencher(A, aA);
    matriz_preencher(B, aA);
    printf("Matriz A\n");
    matriz_exibir(A, aA);
    printf("Matriz B\n");
    matriz_exibir(B, aA);
    multiplicar(A, B, aA, lA, lB, C);
    printf("RESULTADO\n");
    matriz_exibir(C, aA);

    hipDeviceReset();
    exit(EXIT_SUCCESS);
}