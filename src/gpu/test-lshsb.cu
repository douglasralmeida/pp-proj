#include "hip/hip_runtime.h"
/*
** Projeto de Algoritmos Paralelos
** LSH Superbit - TESTE
*/

#include <ctime>
#include <iomanip>
#include <iostream>
#include <random>
#include "math.hpp"
#include "array.hpp"
#include "lsh_superbit.hpp"

#define ARRAY_SIZE 100
#define ARRAY_COUNT 100000
//define ARRAY_SIZE 3
//efine ARRAY_COUNT 100

using namespace std;

int main() {
    double** mm;
    std::default_random_engine generator(time(NULL));
    std::normal_distribution<long double> distribution(0.0, 1.0);

    cout << "LSH SUPERBIT" << endl;
    cout << "============" << endl << endl;

    int stages = 100; //2
    int buckets = 100;//4

    cout << "Gerando entradas aleatorias..." << endl;
    mm = new double*[ARRAY_COUNT];
    for (long i = 0; i < ARRAY_COUNT; i++) {
        mm[i] = new double[ARRAY_SIZE];
        for (int j = 0; j < ARRAY_SIZE; j++)
          mm[i][j] = distribution(generator);  
    }
    int* tables = new int[buckets];
    for (int i = 0; i < buckets; i++)
        tables[i] = 0;

    //começa a medir o tempo aqui
    clock_t begin = clock();

    LSH_Superbit* lsh = new LSH_Superbit(buckets, stages, ARRAY_SIZE);
    cout << "Processando entradas..." << endl;
    for (long i = 0; i < ARRAY_COUNT; i++) {
        for (long j = 0; j < ARRAY_SIZE; j++) {
            //if (mm[i][j] >= 0.0)
            //    cout << ' ';
            //cout << fixed << setprecision(4) << mm[i][j] << '\t';
        }
        lsh->hash(i, mm[i]);
        delete mm[i];
    }

    //termina aqui
    clock_t end = clock();
    double elapsed_secs = double(end - begin) / CLOCKS_PER_SEC;

    cout << endl << "Distribution: ";
    lsh->showCounts();

    cout << endl << "Tempo gasto: " << elapsed_secs << endl << endl;
    
    cout << endl << "Finalizando..." << endl;

    delete tables;
    delete mm;
    delete lsh;
	exit(EXIT_SUCCESS);
}