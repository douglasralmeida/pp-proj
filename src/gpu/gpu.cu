/*
** Projeto de Algoritmos Paralelos
** Objeto GPU
*/

#include <iostream>
#include <hip/hip_runtime_api.h>

#define GPU_ERROR "Erro de processamento na GPU:";

__host__ GPU::GPU() {
    int count;
    hipError_t error;

    cout << "Inicializando GPU..." << endl;
    hipProfilerStart();
    error = hipGetDeviceCount(&count);
    if (error != hipSuccess)
        exitWithFailure();
    cout << "Numero de dispositivos disponíveis: " << count << endl;
    error = hipGetDeviceProperties(&deviceProp, 0);
    if (error != hipSuccess) 
        exitWithFailure();
    cout << "Dispositivo 0 tem a capacidade computacional " << deviceProp.major << '.' << deviceProp.minor << endl;
}

__host__ GPU::~GPU() {
    hipDeviceReset();
}

__host__ void GPU::exitWithFailure() {
    cout << GPU_ERROR << ' ' << hipGetErrorString(erro) << endl;
    exit(EXIT_FAILURE);
}