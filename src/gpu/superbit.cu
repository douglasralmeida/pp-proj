#include "hip/hip_runtime.h"
/*
** Projeto de Algoritmos Paralelos
** Algoritmo Superbit - Implementação CUDA
*/

#include <cmath>
#include <random>
#include <ctime>
#include "array.hpp"
#include "math.hpp"
#include "superbit.hpp"

Superbit::Superbit(const int _dimensions, int _superbit, int _length, int _seed):dimensions(_dimensions) {
    hpbuilder_t builderdata;

    if (_superbit < 1 || _superbit > _dimensions) {
        cout << "Erro. Profundidade N do SuperBit deve estar entre 1 e o número de dimensões." << endl;
        exit(EXIT_FAILURE);
    }
    hyperp_length = _superbit * _length;
    hyperplanes = Array::alloc2d(hyperp_length, dimensions);
    builderdata.v = Array::alloc2d(hyperp_length, dimensions);
    builderdata.superbit = _superbit;
    builderdata.length = _length;
    builderdata.seed = _seed;
    builderdata.w = hyperplanes;
    buildHyperplanes(&builderdata);
    Array::dealloc2d(&builderdata.v);
}

Superbit::Superbit(const int _dimensions, int _superbit, int _length):
    Superbit(_dimensions, _superbit, _length, time(NULL)) {}

Superbit::~Superbit() {
    Array::dealloc2d(&hyperplanes);
}

void Superbit::buildHyperplanes(hpbuilder_t *builderdata) {
    long i, j, k;
    std::default_random_engine generator(builderdata->seed);
    std::normal_distribution<long double> distribution(0.0, 1.0);
    double* v = builderdata->v;
    double* w = builderdata->w;

    for (i = 0; i < hyperp_length; i++) {
        long x = i * dimensions;
        for (j = 0; j < dimensions; j++)
            v[x + j] = distribution(generator);
        Math::normalize(v + x, dimensions);
    }

    for (i = 0; i <= (builderdata->length-1); i++) {
        for (j = 1; j <= builderdata->superbit; j++) {
            int pos = (i * builderdata->superbit + j - 1)*dimensions;
            Array::copy(v + pos, w + pos, dimensions);
            for (k = 1; k <= (j-1); k++) {
                int wpos = (i * builderdata->superbit + k - 1)*dimensions;
                Math::sub(w + pos,
                    Math::product(
                        Math::dotProduct(w + wpos, v + pos, dimensions),
                        w + wpos,
                        dimensions),
                    dimensions);
            }
            Math::normalize(w + pos, dimensions);
        }
    }
}

bool* Superbit::computeSignature(double* v) {
    long pos;
    bool* sig = new bool[hyperp_length];

    for (int i = 0; i < hyperp_length; i++) {
        pos = i * dimensions;
        sig[i] = (Math::dotProduct(hyperplanes + pos, v, dimensions) >= 0.0);
    }

    return sig;
}

int Superbit::getSignatureLength() {
    return hyperp_length;
}

double Superbit::similarity(bool* s1, bool* s2) {
    double sum = 0;

    for (int i = 0; i < hyperp_length; i++)
        if (s1[i] == s2[i])
            sum++;
    sum /= hyperp_length;

    return cos((1 - sum) * acos(-1)); //acos(-1)=pi
}