#include "hip/hip_runtime.h"
/*
** Projeto de Algoritmos Paralelos
** Algoritmo Superbit - Implementação em CPU
*/

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include <cmath>
#include <random>
#include <ctime>
#include "array.hpp"
#include "math.hpp"
#include "superbit.hpp"

#define THREADS 64
#define BLOCKS 64

__global__ void cuda_rand_init(unsigned long seed, hiprandState* state) {
    int id = threadIdx.x + blockIdx.x * blockDim.x;

    hiprand_init(seed, id, 0, &state[id]);
}

__global__ void cuda_distribuition(hiprandState* state, double* vector) {
    int id = threadIdx.x + blockIdx.x * blockDim.x;

    vector[id] = hiprand_normal_double(&state[id]);
}

Superbit::Superbit(const int _dimensions, int _superbit, long _length, int _seed):dimensions(_dimensions) {
    hpbuilder_t builderdata;

    if (_superbit < 1 || _superbit > _dimensions) {
        cout << "Erro. Profundidade N do SuperBit deve estar entre 1 e o número de dimensões." << endl;
        exit(EXIT_FAILURE);
    }
    hyperp_length = _superbit * _length;
    hyperplanes = Array::alloc2d(hyperp_length, dimensions);
    builderdata.v = Array::alloc2d(hyperp_length, dimensions);
    builderdata.superbit = _superbit;
    builderdata.length = _length;
    builderdata.seed = _seed;
    builderdata.w = hyperplanes;
    buildHyperplanes(&builderdata);
    Array::dealloc2d(&builderdata.v);
}

Superbit::Superbit(const int _dimensions, int _superbit, long _length):
    Superbit(_dimensions, _superbit, _length, time(NULL)) {}

Superbit::~Superbit() {
    Array::dealloc2d(&hyperplanes);
}

void Superbit::buildHyperplanes(hpbuilder_t *builderdata) {
    long i, j, k;
    hiprandState *devStates = hipMalloc((void**)devStates, THREADS * BLOCKS * sizeof(hiprandState));
    double* v = builderdata->v;
    double* w = builderdata->w;

    cuda_distribuition<<<BLOCKS, THREADS>>>(devStates, v);

    //Normaliza
    std::cout << v[10] << endl;

    hipFree(v);
    exit(0);


    for (i = 0; i <= (builderdata->length-1); i++) {
        for (j = 1; j <= builderdata->superbit; j++) {
            int pos = (i * builderdata->superbit + j - 1)*dimensions;
            Array::copy(v + pos, w + pos, dimensions);
            for (k = 1; k <= (j-1); k++) {
                int wpos = (i * builderdata->superbit + k - 1)*dimensions;
                Math::sub(w + pos,
                    Math::product(
                        Math::dotProduct(w + wpos, v + pos, dimensions),
                        w + wpos,
                        dimensions),
                    dimensions);
            }
            Math::normalize(w + pos, dimensions);
        }
    }
}

bool* Superbit::computeSignature(double* v) {
    long pos;
    bool* sig = new bool[hyperp_length];

    for (long i = 0; i < hyperp_length; i++) {
        pos = i * dimensions;
        sig[i] = (Math::dotProduct(hyperplanes + pos, v, dimensions) >= 0.0);
    }

    return sig;
}

long Superbit::getSignatureLength() {
    return hyperp_length;
}

double Superbit::similarity(bool* s1, bool* s2) {
    double sum = 0;

    for (long i = 0; i < hyperp_length; i++)
        if (s1[i] == s2[i])
            sum++;
    sum /= hyperp_length;

    return cos((1 - sum) * acos(-1)); //acos(-1)=pi
}